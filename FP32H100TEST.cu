#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <vector>
#include <string>
#include <cctype>
#include <cmath>
#include <map>
#include <cstdint>

// 操作码枚举
enum Opcode {
    ADD, SUB, MUL, FMA, FMS, FNMA, FNMS,
    CMPEQ, CMPLT, CMPLE, CMPGT,
    CMPLTNUM, CMPLENUM, CMPGTNUM, UNORDERED
};

// 舍入模式枚举
enum RoundMode {
    RND_ZERO, RND_MINUS_INF, RND_PLUS_INF, RND_NEAREST
};

// 测试用例结构
struct TestCase {
    Opcode opcode;
    RoundMode roundMode;
    uint32_t operandA;
    uint32_t operandB;
    uint32_t operandC;
};

// 结果结构
struct Result {
    uint32_t result;
};

// 字符串到操作码映射
std::map<std::string, Opcode> opcodeMap = {
    {"ADD", ADD}, {"SUB", SUB}, {"MUL", MUL}, {"FMA", FMA}, {"FMS", FMS},
    {"FNMA", FNMA}, {"FNMS", FNMS}, {"CMPEQ", CMPEQ}, {"CMPLT", CMPLT},
    {"CMPLE", CMPLE}, {"CMPGT", CMPGT}, {"CMPLTNUM", CMPLTNUM},
    {"CMPLENUM", CMPLENUM}, {"CMPGTNUM", CMPGTNUM}, {"UNORDERED", UNORDERED}
};

// 字符串到舍入模式映射
std::map<std::string, RoundMode> roundModeMap = {
    {"RND_ZERO", RND_ZERO}, {"RND_MINUS_INF", RND_MINUS_INF},
    {"RND_PLUS_INF", RND_PLUS_INF}, {"RND_NEAREST", RND_NEAREST}
};

// CUDA内核：执行测试用例（仅原始计算）
__global__ void executeTests(const TestCase* testCases, Result* results, int numTests) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numTests) return;
    
    TestCase tc = testCases[idx];
    float a = __uint_as_float(tc.operandA);
    float b = __uint_as_float(tc.operandB);
    float c = __uint_as_float(tc.operandC);
    float res = 0.0f;
    
    // 直接硬件计算（无异常检测）
    switch (tc.opcode) {
        case ADD:
            if (tc.roundMode == RND_ZERO) res = __fadd_rz(a, c);
            else if (tc.roundMode == RND_MINUS_INF) res = __fadd_rd(a, c);
            else if (tc.roundMode == RND_PLUS_INF) res = __fadd_ru(a, c);
            else res = __fadd_rn(a, c);
            break;
        case SUB:
            if (tc.roundMode == RND_ZERO) res = __fsub_rz(a, c);
            else if (tc.roundMode == RND_MINUS_INF) res = __fsub_rd(a, c);
            else if (tc.roundMode == RND_PLUS_INF) res = __fsub_ru(a, c);
            else res = __fsub_rn(a, c);
            break;
        case MUL:
            if (tc.roundMode == RND_ZERO) res = __fmul_rz(a, b);
            else if (tc.roundMode == RND_MINUS_INF) res = __fmul_rd(a, b);
            else if (tc.roundMode == RND_PLUS_INF) res = __fmul_ru(a, b);
            else res = __fmul_rn(a, b);
            break;
        case FMA:
            if (tc.roundMode == RND_ZERO) res = __fmaf_rz(a, b, c);
            else if (tc.roundMode == RND_MINUS_INF) res = __fmaf_rd(a, b, c);
            else if (tc.roundMode == RND_PLUS_INF) res = __fmaf_ru(a, b, c);
            else res = __fmaf_rn(a, b, c);
            break;
        case FMS:
            if (tc.roundMode == RND_ZERO) res = __fmaf_rz(a, b, -c);
            else if (tc.roundMode == RND_MINUS_INF) res = __fmaf_rd(a, b, -c);
            else if (tc.roundMode == RND_PLUS_INF) res = __fmaf_ru(a, b, -c);
            else res = __fmaf_rn(a, b, -c);
            break;
        case FNMA:
            if (tc.roundMode == RND_ZERO) res = __fmaf_rz(-a, b, c);
            else if (tc.roundMode == RND_MINUS_INF) res = __fmaf_rd(-a, b, c);
            else if (tc.roundMode == RND_PLUS_INF) res = __fmaf_ru(-a, b, c);
            else res = __fmaf_rn(-a, b, c);
            break;
        case FNMS:
            if (tc.roundMode == RND_ZERO) res = __fmaf_rz(-a, b, -c);
            else if (tc.roundMode == RND_MINUS_INF) res = __fmaf_rd(-a, b, -c);
            else if (tc.roundMode == RND_PLUS_INF) res = __fmaf_ru(-a, b, -c);
            else res = __fmaf_rn(-a, b, -c);
            break;
        case CMPEQ:
            res = (a == c) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
        case CMPLT:
            res = (a < c) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
        case CMPLE:
            res = (a <= c) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
        case CMPGT:
            res = (a > c) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
        case CMPLTNUM:
            if (isnan(a)) {
                res = a;
            } else if (isnan(c)) {
                res = c;
            } else {
                res = (a < c) ? a : c;
            }
            break;
        case CMPLENUM:
            if (isnan(a)) {
                res = a;
            } else if (isnan(c)) {
                res = c;
            } else {
                res = (a <= c) ? a : c;
            }
            break;
        case CMPGTNUM:
            if (isnan(a)) {
                res = a;
            } else if (isnan(c)) {
                res = c;
            } else {
                res = (a > c) ? a : c;
            }
            break;
        case UNORDERED:
            res = (isnan(a) || isnan(c)) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
    }
    
    // 存储原始结果
    results[idx].result = __float_as_uint(res);
}

// 解析十六进制字符串
uint32_t parseHex(const std::string& hexStr) {
    return std::stoul(hexStr, nullptr, 16);
}

// 读取输入文件
std::vector<TestCase> readInputFile(const std::string& filename) {
    std::vector<TestCase> testCases;
    std::ifstream file(filename);
    std::string line;
    
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::string token;
        std::vector<std::string> tokens;
        
        while (std::getline(iss, token, ',')) {
            token.erase(0, token.find_first_not_of(' '));
            token.erase(token.find_last_not_of(' ') + 1);
            tokens.push_back(token);
        }
        
        if (tokens.size() == 5) {
            TestCase tc;
            tc.opcode = opcodeMap[tokens[0]];
            tc.roundMode = roundModeMap[tokens[1]];
            tc.operandA = parseHex(tokens[2]);
            tc.operandB = parseHex(tokens[3]);
            tc.operandC = parseHex(tokens[4]);
            testCases.push_back(tc);
        }
    }
    
    return testCases;
}

// 写输出文件（简化版）
void writeOutputFile(const std::string& filename, 
                    const std::vector<TestCase>& testCases,
                    const std::vector<Result>& results) {
    std::ofstream file(filename);
    file << "Opcode, Rnd, Operand A, Operand B, Operand C, Result\n";
    
    // 反向映射用于输出
    std::map<Opcode, std::string> opcodeStr;
    for (const auto& p : opcodeMap) opcodeStr[p.second] = p.first;
    
    std::map<RoundMode, std::string> roundModeStr;
    for (const auto& p : roundModeMap) roundModeStr[p.second] = p.first;
    
    for (size_t i = 0; i < testCases.size(); ++i) {
        const TestCase& tc = testCases[i];
        const Result& res = results[i];
        
        file << opcodeStr[tc.opcode] << ", "
             << roundModeStr[tc.roundMode] << ", "
             << "0x" << std::hex << std::setw(8) << std::setfill('0') << tc.operandA << ", "
             << "0x" << std::hex << std::setw(8) << std::setfill('0') << tc.operandB << ", "
             << "0x" << std::hex << std::setw(8) << std::setfill('0') << tc.operandC << ", "
             << "0x" << std::hex << std::setw(8) << std::setfill('0') << res.result << "\n";
    }
}

int main() {
    // 读取输入文件
    std::vector<TestCase> testCases = readInputFile("input.txt");
    int numTests = testCases.size();
    
    // 分配设备内存
    TestCase* d_testCases;
    Result* d_results;
    hipMalloc(&d_testCases, numTests * sizeof(TestCase));
    hipMalloc(&d_results, numTests * sizeof(Result));
    
    // 复制数据到设备
    hipMemcpy(d_testCases, testCases.data(), numTests * sizeof(TestCase), hipMemcpyHostToDevice);
    
    // 启动内核
    int blockSize = 512;
    int gridSize = (numTests + blockSize - 1) / blockSize;
    executeTests<<<gridSize, blockSize>>>(d_testCases, d_results, numTests);
    hipDeviceSynchronize();
    
    // 复制结果回主机
    std::vector<Result> results(numTests);
    hipMemcpy(results.data(), d_results, numTests * sizeof(Result), hipMemcpyDeviceToHost);
    
    // 写输出文件
    writeOutputFile("h100test_output.txt", testCases, results);
    
    // 清理
    hipFree(d_testCases);
    hipFree(d_results);
    
    std::cout << "H100 FP32 测试完成，结果已写入 h100test_output.txt" << std::endl;
    return 0;
}